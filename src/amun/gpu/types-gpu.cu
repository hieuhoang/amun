#include <iostream>
#include "types-gpu.h"
#include "mblas/handles.h"

using namespace std;

namespace amunmt {
namespace GPU {

void HandleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    std::cerr << "ERROR: " << hipGetErrorString(err) << " in " << file << " at line " << line << std::endl;
    abort();
    //exit( EXIT_FAILURE );
  }
}

void HandleErrorCublas(hipblasStatus_t err, const char *file, int line ) {
  if (err != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS ERROR: " << err << " in " << file << " at line " << line << std::endl;
    abort();
    //exit( EXIT_FAILURE );
  }
}

}
}
