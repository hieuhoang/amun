#include "npz_converter.h"
#include "common/exception.h"
#include "mblas/tensor_functions.h"

using namespace std;

namespace amunmt {
namespace GPU {

NpzConverter::NpzConverter(const std::string& file)
  : model_(cnpy::npz_load(file)),
    destructed_(false)
{
}

NpzConverter::~NpzConverter() {
  if(!destructed_)
    model_.destruct();
}

void NpzConverter::Destruct() {
  model_.destruct();
  destructed_ = true;
}

template<typename T>
T Debug(const T *data, unsigned size)
{
  T sum = 0;
  for (unsigned i = 0; i < size; ++i) {
    sum += data[i];
  }
  return sum;
}

std::shared_ptr<mblas::Tensor> NpzConverter::get(const std::string& key, bool mandatory, bool transpose) const
{
  //mblas::TestMemCpy();

  std::shared_ptr<mblas::Tensor> ret;
  auto it = model_.find(key);
  if(it != model_.end()) {
    NpyMatrixWrapper np(it->second);
    unsigned size = np.size();

    mblas::Tensor *matrix = new mblas::Tensor(np.size1(), np.size2(), 1, 1);
    mblas::copy(np.data(), size, matrix->data(), hipMemcpyHostToDevice);

    if (transpose) {
      mblas::Transpose(*matrix);
    }

    ret.reset(matrix);
  }
  else if (mandatory) {
    std::cerr << "Error: Tensor not found:" << key << std::endl;
    //amunmt_UTIL_THROW2(strm.str()); //  << key << std::endl
    abort();
  }
  else {
    mblas::Tensor *matrix = new mblas::Tensor();
    ret.reset(matrix);
  }

  //std::cerr << "key=" << key << " " << ret->Debug(1) << std::endl;
  return ret;
}

std::shared_ptr<mblas::Tensor> NpzConverter::getFirstOfMany(const std::vector<std::pair<std::string, bool>> keys, bool mandatory) const
{
  std::shared_ptr<mblas::Tensor> ret;
  for (auto key : keys) {
    auto it = model_.find(key.first);
    if(it != model_.end()) {
      NpyMatrixWrapper np(it->second);
      mblas::Tensor *matrix = new mblas::Tensor(np.size1(), np.size2(), 1, 1);
      mblas::copy(np.data(), np.size(), matrix->data(), hipMemcpyHostToDevice);

      if (key.second) {
        mblas::Transpose(*matrix);
      }
      ret.reset(matrix);
      return ret;
    }
  }

  if (mandatory) {
    std::cerr << "Error: Tensor not found:" << keys[0].first << std::endl;
    //amunmt_UTIL_THROW2(strm.str()); //  << key << std::endl
    abort();
  }
  else {
    std::cerr << "Optional matrix not found, continuing: " << keys[0].first << std::endl;
  }

  return ret;

}
  

}
}
