#include "best_hyps.h"
#include "common/beam.h"
#include "common/history.h"
#include "common/histories.h"

using namespace std;

namespace amunmt {
namespace GPU {

BestHyps::BestHyps(const God &god, unsigned maxBeamSize)
      : BaseBestHyps(god, maxBeamSize),
        keys_(god.Get<unsigned>("beam-size") * god.Get<unsigned>("mini-batch")),
        costs_(god.Get<unsigned>("beam-size") * god.Get<unsigned>("mini-batch")),
        maxBeamSize_(god.Get<unsigned>("beam-size"))
{
  if (!god_.UseFusedSoftmax()) {
    NthElement *obj = new NthElement(god.Get<unsigned>("beam-size"), god.Get<unsigned>("mini-batch"));
    nthElement_.reset(obj);
  }
}

void BestHyps::BeginSentenceState(unsigned batchSize)
{
  beamSizes_.clear();
  beamSizes_.resize(batchSize, 1);
}

const void* BestHyps::GetBeamSizes() const
{
  return &beamSizes_;
}

void BestHyps::DisAllowUNK(mblas::Tensor& Prob) {
  SetColumn(Prob, UNK_ID, std::numeric_limits<float>::lowest());
}

void BestHyps::FindBests(mblas::Tensor& Probs,
                         std::vector<float>& outCosts,
                         std::vector<unsigned>& outKeys,
                         const bool isFirst)
{
  nthElement_->getNBestList(beamSizes_, Probs, outCosts, outKeys, isFirst);
}

// fast fused softmax and nth_element
void BestHyps::FindBests(mblas::Tensor& Probs,
                         mblas::Vector<NthOutBatch> &nBest,
                         std::vector<float>& outCosts,
                         std::vector<unsigned>& outKeys,
                         const bool isFirst)
{
  getNBestList(Probs, nBest, outCosts, outKeys, isFirst);
}

std::vector<SoftAlignmentPtr> BestHyps::GetAlignments(const std::vector<ScorerPtr>& scorers,
                                            unsigned hypIndex)
{
  std::vector<SoftAlignmentPtr> alignments;
  for (auto& scorer : scorers) {
    if (GPU::EncoderDecoder* encdec = dynamic_cast<GPU::EncoderDecoder*>(scorer.get())) {
      const mblas::Tensor &attention = encdec->GetAttention();
      unsigned attLength = attention.dim(1);

      SoftAlignment *softAlignment = new SoftAlignment(attLength);
      mblas::copy(
          attention.data() + hypIndex * attLength,
          attLength,
          softAlignment->data(),
          hipMemcpyDeviceToHost
      );

      alignments.emplace_back(softAlignment);
    } else {
      amunmt_UTIL_THROW2("Return Alignment is allowed only with Nematus scorer.");
    }
  }
  return alignments;
}

// standard nth_element
void  BestHyps::CalcBeam(
    const Beam& prevHyps,
    const std::vector<ScorerPtr>& scorers,
    const Words& filterIndices,
    std::vector<Beam>& beams)
{
  BEGIN_TIMER("CalcBeam");

  using namespace mblas;

  mblas::Tensor& Probs = static_cast<mblas::Tensor&>(scorers[0]->GetProbs());

  std::vector<float> vCosts;
  for (auto& h : prevHyps) {
    vCosts.push_back(h->GetCost());
  }
  const bool isFirst = (vCosts[0] == 0.0f) ? true : false;

  if (isFirst) {
    for (auto& beamSize : beamSizes_) {
      beamSize = maxBeamSize_;
    }
  }

  mblas::copy(vCosts.data(),
              vCosts.size(),
              costs_.data(),
              hipMemcpyHostToDevice);
  //mblas::copy(vCosts.begin(), vCosts.end(), costs_.begin());

  unsigned beamSizeSum = std::accumulate(beamSizes_.begin(), beamSizes_ .end(), 0);

  std::vector<float> bestCosts;
  std::vector<unsigned> bestKeys;


  if (god_.UseFusedSoftmax()) {
    const mblas::Tensor& b4 = *static_cast<const mblas::Tensor*>(scorers[0]->GetBias());
    mblas::Vector<NthOutBatch> &nBest = *static_cast<mblas::Vector<NthOutBatch>*>(scorers[0]->GetNBest());
    nBest.newSize(beamSizeSum);

    bool requireProb = maxBeamSize_ > 1 || god_.Get<bool>("n-best");
    //cerr << "doSoftmax=" << doSoftmax << endl;

    BEGIN_TIMER("GetProbs.LogSoftmaxAndNBest");
    mblas::LogSoftmaxAndNBest(nBest, Probs, b4, costs_, forbidUNK_, maxBeamSize_, beamSizes_, beamSizeSum, isFirst, requireProb);
    PAUSE_TIMER("GetProbs.LogSoftmaxAndNBest");
    //std::cerr << "2Probs=" << Probs.Debug(1) << std::endl;

    FindBests(Probs, nBest, bestCosts, bestKeys, isFirst);
  }
  else {
    BroadcastVecColumn(weights_.at(scorers[0]->GetName()) * _1 + _2, Probs, costs_);

    for (unsigned i = 1; i < scorers.size(); ++i) {
      mblas::Tensor &currProbs = static_cast<mblas::Tensor&>(scorers[i]->GetProbs());

      Element(_1 + weights_.at(scorers[i]->GetName()) * _2, Probs, currProbs);
    }

    if (forbidUNK_) {
      DisAllowUNK(Probs);
    }

    FindBests(Probs, bestCosts, bestKeys, isFirst);
  }

  std::vector<std::vector<float>> breakDowns;
  if (god_.ReturnNBestList()) {
      breakDowns.push_back(bestCosts);
      for (unsigned i = 1; i < scorers.size(); ++i) {
        std::vector<float> modelCosts(beamSizeSum);
        mblas::Tensor &currProbs = static_cast<mblas::Tensor&>(scorers[i]->GetProbs());

        nthElement_->getValueByKey(modelCosts, currProbs);
        breakDowns.push_back(modelCosts);
      }
  }

  std::map<unsigned, unsigned> batchMap;
  unsigned tmp = 0;
  for (unsigned batchID = 0; batchID < beamSizes_.size(); ++batchID) {
    for (unsigned t = 0; t < beamSizes_[batchID]; ++t) {
      batchMap[tmp++] = batchID;
    }
  }

  for (unsigned i = 0; i < beamSizeSum; i++) {
    unsigned wordIndex = bestKeys[i] % Probs.dim(1);
    if (isInputFiltered_) {
      wordIndex = filterIndices[wordIndex];
    }

    unsigned hypIndex  = bestKeys[i] / Probs.dim(1);
    float cost = bestCosts[i];

    HypothesisPtr hyp;
    if (returnAttentionWeights_) {
      hyp.reset(new Hypothesis(prevHyps[hypIndex], wordIndex, hypIndex, cost,
                               GetAlignments(scorers, hypIndex)));
    } else {
      hyp.reset(new Hypothesis(prevHyps[hypIndex], wordIndex, hypIndex, cost));
    }

    //cerr << "god_.ReturnNBestList()=" << god_.ReturnNBestList() << endl;
    if(god_.ReturnNBestList()) {
      hyp->GetCostBreakdown().resize(scorers.size());
      float sum = 0;
      for (unsigned j = 0; j < scorers.size(); ++j) {
        if (j == 0)
          hyp->GetCostBreakdown()[0] = breakDowns[0][i];
        else {
          float cost = 0;
          if (j < scorers.size()) {
              if (prevHyps[hypIndex]->GetCostBreakdown().size() < scorers.size())
                const_cast<HypothesisPtr&>(prevHyps[hypIndex])->GetCostBreakdown().resize(scorers.size(), 0.0f);
              cost = breakDowns[j][i] + const_cast<HypothesisPtr&>(prevHyps[hypIndex])->GetCostBreakdown()[j];
          }
          sum += weights_.at(scorers[j]->GetName()) * cost;
          hyp->GetCostBreakdown()[j] = cost;
        }
      }
      hyp->GetCostBreakdown()[0] -= sum;
      hyp->GetCostBreakdown()[0] /= weights_.at(scorers[0]->GetName());
    }

    beams[batchMap[i]].push_back(hyp);
  }

  PAUSE_TIMER("CalcBeam");
}

//////////////////////////////////////////////////////////////////////////
void BestHyps::getNBestList(
                  mblas::Tensor& Probs,
                  mblas::Vector<NthOutBatch> &nBest,
                  std::vector<float>& outCosts,
                  std::vector<unsigned>& outKeys,
                  const bool isFirst) const
{
  GetPairs(nBest, outKeys, outCosts);
  assert(outCosts.size() == outKeys.size());

  /*
  cerr << "outCosts/outKeys=";
  for (unsigned i = 0; i < outKeys.size(); ++i) {
    cerr << "(" << outCosts[i] << "," << outKeys[i] << ") ";
  }
  cerr << endl;
  */
  //cerr << endl;
}

void BestHyps::GetPairs(mblas::Vector<NthOutBatch> &nBest,
              std::vector<unsigned>& outKeys,
              std::vector<float>& outValues) const
{
  //cerr << "top=" << top2.size() << " nBest=" << nBest.size() << endl;
  outKeys.resize(nBest.size());
  outValues.resize(nBest.size());

  std::vector<NthOutBatch> hostVec(nBest.size());
  mblas::copy(nBest.data(), nBest.size(), hostVec.data(), hipMemcpyDeviceToHost);

  for (unsigned i = 0; i < nBest.size(); ++i) {
    outKeys[i] = hostVec[i].ind;
    outValues[i] = hostVec[i].score;
  }
}

bool BestHyps::CalcBeam(
    const std::vector<ScorerPtr>& scorers,
    const Words &filterIndices,

    std::shared_ptr<Histories>& histories,
    Beam& prevHyps,
    States& states,
    States& nextStates,
    unsigned decoderStep)
{
    unsigned batchSize = beamSizes_.size();
    Beams beams(batchSize);
    CalcBeam(prevHyps, scorers, filterIndices, beams);
    histories->Add(beams);

    //cerr << "batchSize=" << batchSize << endl;
    histories->SetActive(false);
    Beam survivors;
    for (unsigned batchId = 0; batchId < batchSize; ++batchId) {
      const History &hist = *histories->at(batchId);
      unsigned maxLength = hist.GetMaxLength();

      //cerr << "beamSizes[batchId]=" << batchId << " " << beamSizes[batchId] << " " << maxLength << endl;
      for (auto& h : beams[batchId]) {
        if (decoderStep < maxLength && h->GetWord() != EOS_ID) {
          survivors.push_back(h);

          histories->SetActive(batchId, true);
        } else {
          --beamSizes_[batchId];
        }
      }
    }

    if (survivors.size() == 0) {
      return false;
    }

    for (unsigned i = 0; i < scorers.size(); i++) {
      scorers[i]->AssembleBeamState(*nextStates[i], survivors, *states[i]);
    }

    //cerr << "survivors=" << survivors.size() << endl;
    prevHyps.swap(survivors);
    return true;
}

} // namespace
}
