#include "hip/hip_runtime.h"
#include "encoder.h"
#include "common/sentences.h"

using namespace std;

namespace amunmt {
namespace GPU {

Encoder::Encoder(const Weights& model, const YAML::Node& config)
  : embeddings_(model.encEmbeddings_),
    forwardRnn_(InitForwardCell(model, config)),
    backwardRnn_(InitBackwardCell(model, config))
{}

std::unique_ptr<Cell> Encoder::InitForwardCell(const Weights& model, const YAML::Node& config){
  std::string celltype = config["enc-cell"] ? config["enc-cell"].as<std::string>() : "gru";
  if (celltype == "lstm") {
    return unique_ptr<Cell>(new LSTM<Weights::EncForwardLSTM>(*(model.encForwardLSTM_)));
  } else if (celltype == "mlstm") {
    return unique_ptr<Cell>(new Multiplicative<LSTM, Weights::EncForwardLSTM>(*model.encForwardMLSTM_));
  } else if (celltype == "gru") {
    return unique_ptr<Cell>(new GRU<Weights::EncForwardGRU>(*(model.encForwardGRU_)));
  }

  assert(false);
  return unique_ptr<Cell>(nullptr);
}

std::unique_ptr<Cell> Encoder::InitBackwardCell(const Weights& model, const YAML::Node& config){
  std::string enccell = config["enc-cell"] ? config["enc-cell"].as<std::string>() : "gru";
  std::string celltype = config["enc-cell-r"] ? config["enc-cell-r"].as<std::string>() : enccell;
  if (celltype == "lstm") {
    return unique_ptr<Cell>(new LSTM<Weights::EncBackwardLSTM>(*(model.encBackwardLSTM_)));
  } else if (celltype == "mlstm") {
    return unique_ptr<Cell>(new Multiplicative<LSTM, Weights::EncBackwardLSTM>(*model.encBackwardMLSTM_));
  } else if (celltype == "gru") {
    return unique_ptr<Cell>(new GRU<Weights::EncBackwardGRU>(*(model.encBackwardGRU_)));
  }

  assert(false);
  return unique_ptr<Cell>(nullptr);
}

unsigned GetMaxLength(const Sentences& source, unsigned tab) {
  unsigned maxLength = source.Get(0).GetWords(tab).size();
  for (unsigned i = 0; i < source.size(); ++i) {
    const Sentence &sentence = source.Get(i);
    maxLength = std::max(maxLength, (unsigned) sentence.GetWords(tab).size());
  }
  return maxLength;
}

std::vector<std::vector<FactWord>> GetBatchInput(const Sentences& source, unsigned tab, unsigned maxLen) {
  std::vector<std::vector<FactWord>> matrix(maxLen, std::vector<FactWord>(source.size()));

  for (unsigned batchIdx = 0; batchIdx < source.size(); ++batchIdx) {
    for (unsigned wordIdx = 0; wordIdx < source.Get(batchIdx).GetFactors(tab).size(); ++wordIdx) {
        matrix[wordIdx][batchIdx] = source.Get(batchIdx).GetFactors(tab)[wordIdx];
    }
  }

  return matrix;
}

void Encoder::Encode(const Sentences& source,
                      unsigned tab,
                      mblas::Tensor& context,
                      std::vector<unsigned> &h_sentenceLengths,
                      mblas::Vector<unsigned> &sentenceLengths)
{
  unsigned maxSentenceLength = GetMaxLength(source, tab);

  h_sentenceLengths.resize(source.size());
  sentenceLengths.newSize(source.size());

  for (unsigned i = 0; i < source.size(); ++i) {
    h_sentenceLengths[i] = source.Get(i).GetWords(tab).size();
  }

  mblas::copy(h_sentenceLengths.data(),
              h_sentenceLengths.size(),
              sentenceLengths.data(),
              hipMemcpyHostToDevice);

  //cerr << "GetContext1=" << context.Debug(1) << endl;
  context.NewSize(maxSentenceLength,
                 forwardRnn_.GetStateLength().output + backwardRnn_.GetStateLength().output,
                 1,
                 source.size());
  //cerr << "GetContext2=" << context.Debug(1) << endl;

  auto input = GetBatchInput(source, tab, maxSentenceLength);

  for (unsigned i = 0; i < input.size(); ++i) {
    if (i >= embeddedWords_.size()) {
      embeddedWords_.emplace_back();
    }
    embeddings_.Lookup(embeddedWords_[i], input[i]);
    //cerr << "embeddedWord_=" << embeddedWords_[i].Debug(1) << endl;
  }

  //cerr << "GetContext3=" << context.Debug(1) << endl;
  forwardRnn_.Encode(embeddedWords_.cbegin(),
                         embeddedWords_.cbegin() + maxSentenceLength,
                         context, source.size(), false);
  //cerr << "GetContext4=" << context.Debug(1) << endl;

  backwardRnn_.Encode(embeddedWords_.crend() - maxSentenceLength,
                          embeddedWords_.crend() ,
                          context, source.size(), true, &sentenceLengths);
  //cerr << "GetContext5=" << context.Debug(1) << endl;
}

}
}

