#include "handles.h"
#include "gpu/types-gpu.h"

namespace amunmt {
namespace GPU {
namespace mblas {

CudaStreamHandler::CudaStreamHandler()
{
  HANDLE_ERROR( hipStreamCreate(&stream_));
  // hipStreamCreateWithFlags(stream_.get(), hipStreamNonBlocking);
}

CudaStreamHandler::~CudaStreamHandler()
{
  HANDLE_ERROR(hipStreamDestroy(stream_));
}

/////////////////////////////////////////////////////////////////////////////////////////

CublasHandler::CublasHandler()
{
  hipblasStatus_t stat;
  stat = hipblasCreate(&handle_);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("hipblasCreate initialization failed\n");
    abort();
  }

#if CUDA_VERSION >= 9000
  /*
    stat = hipblasSetMathMode(handle_, HIPBLAS_TENSOR_OP_MATH);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      printf ("hipblasSetMathMode failed\n");
      abort();
    }
  */
#endif
		  
  stat = hipblasSetStream(handle_, CudaStreamHandler::GetStream());
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("hipblasSetStream initialization failed\n");
    abort();
  }
}

CublasHandler::~CublasHandler() {
  hipblasDestroy(handle_);
}


} // namespace
}
}
